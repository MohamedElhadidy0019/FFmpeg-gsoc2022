#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, Dummy
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cuda/vector_helpers.cuh"

extern "C" {

__global__ void Process_uchar(hipTextureObject_t src_tex_Y, hipTextureObject_t src_tex_U, hipTextureObject_t src_tex_V,
                              uchar *dst_Y, uchar *dst_U, uchar *dst_V,
                              int width, int height, int pitch,
                              int width_uv, int height_uv, int pitch_uv)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y >= height || x >= width)
        return;
    dst_Y[y*pitch + x] = tex2D<float>(src_tex_Y, x, y) * 255;

    if (y >= height_uv || x >= width_uv)
        return;
    dst_U[y*pitch_uv + x] = tex2D<float>(src_tex_U, x, y) * 255;
    dst_V[y*pitch_uv + x] = tex2D<float>(src_tex_V, x, y) * 255;
}

__global__ void Process_uchar2(hipTextureObject_t src_tex_Y, hipTextureObject_t src_tex_UV, hipTextureObject_t unused1,
                               uchar *dst_Y, uchar2 *dst_UV, uchar *unused2,
                               int width, int height, int pitch,
                               int width_uv, int height_uv, int pitch_uv)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y >= height || x >= width)
        return;
    dst_Y[y*pitch + x] = tex2D<float>(src_tex_Y, x, y) * 255;

    if (y >= height_uv || x >= width_uv)
        return;
    float2 uv = tex2D<float2>(src_tex_UV, x, y) * 255;
    dst_UV[y*pitch_uv + x] = make_uchar2(uv.x, uv.y);
}

}
